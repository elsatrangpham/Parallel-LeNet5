#include "hip/hip_runtime.h"
#ifndef USE_CUDA

#include "filter.h"

#define TILE_WIDTH 8 // GCD(24, 8) in C1 and C3
#define NF (6 * 16 * 25) // in C3 (> C1)
__constant__ float c_d_filter[NF];

#define CHECK(call)                                          \
  {                                                          \
    const hipError_t error = call;                          \
    if (error != hipSuccess)                                \
    {                                                        \
      fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__); \
      fprintf(stderr, "code: %d, reason: %s\n", error,       \
              hipGetErrorString(error));                    \
      exit(EXIT_FAILURE);                                    \
    }                                                        \
  }

__global__ void filter1(float *d_in, int channel_in, int height_in, int width_in,
                        float *d_out, int channel_out, int height_out, int width_out,
                        float *filter, int filterWidth, int W_grid,
                        float *d_bias)
{
  // Indices
  int n = blockIdx.x;                                       // sample index
  int m = blockIdx.y;                                       // channel out index
  int h = (blockIdx.z / W_grid) * blockDim.y + threadIdx.y; // row index in output matrix
  int w = (blockIdx.z % W_grid) * blockDim.x + threadIdx.x; // column index in output matrix

  int sample_strt = height_in * width_in * channel_in * n;      // sample index in input
  int h_in = h + filterWidth / 2;                               // row index in input
  int w_in = w + filterWidth / 2;                               // column index in input
  int filter_strt = channel_in * filterWidth * filterWidth * m; // filter index
  float acc = 0;                                                // pixel conv result
  // output index: (sample index) + (channel index) + (current channel)
  int i_out = (height_out * width_out * channel_out * n) + (height_out * width_out * m) + (h * width_out + w);

  // conv for the pixel in each channel_in
  for (int i_channel = 0; i_channel < channel_in; i_channel++)
  {
    for (int r = h_in - (filterWidth - 1) / 2; r <= h_in + (filterWidth - 1) / 2; r++)
    {
      for (int c = w_in - (filterWidth - 1) / 2; c <= w_in + (filterWidth - 1) / 2; c++)
      {
        // conv
        acc += d_in[r * width_in + c + sample_strt] * filter[filter_strt++];
      }
    }
    // Move to next channel in
    sample_strt += height_in * width_in;
  }
  // Bias adding
  acc += d_bias[m];

  // Final result
  d_out[i_out] = acc;

  // Check input images
  // int d_i = 0;
  // for (int col = 0; col < n; ++col)
  // {
  //   for (int row = 0; row < channel_in * height_in * width_in; ++row)
  //   {
  //     printf("%f ", d_in[d_i++]);
  //   }
  //   printf("\n");
  // }

  // Check input bias
  // for (int i = 0; i < channel_out; ++i)
  //   printf("%f\n", d_bias[i]);
}

int invoke_kernel(const float *h_in, int channel_in, int height_in, int width_in,
                  float *&h_out, int height_out, int width_out, int channel_out,
                  int n_sample, int filter_type, float *h_bias,
                  float *filter, int filterWidth, int stride, int pad_w, int pad_h)
{
  // TODO: Allocate device memories
  float *d_in, *d_out, *d_filter, *d_bias;
  size_t nBytes_d_in = height_in * width_in * channel_in * n_sample * sizeof(float);
  size_t nBytes_d_out = height_out * width_out * channel_out * n_sample * sizeof(float);
  size_t nBytes_d_filter = channel_in * filterWidth * filterWidth * channel_out * sizeof(float);
  size_t nBytes_d_bias = channel_out * sizeof(float);

  CHECK(hipMalloc((void **)&d_in, nBytes_d_in));
  CHECK(hipMalloc((void **)&d_out, nBytes_d_out));
  CHECK(hipMalloc((void **)&d_filter, nBytes_d_filter));
  CHECK(hipMalloc((void **)&d_bias, nBytes_d_bias));

  // TODO: Copy data to device memories
  CHECK(hipMemcpy(d_in, h_in, nBytes_d_in, hipMemcpyHostToDevice));
  CHECK(hipMemcpy(d_filter, filter, nBytes_d_filter, hipMemcpyHostToDevice));
  CHECK(hipMemcpy(d_bias, h_bias, nBytes_d_bias, hipMemcpyHostToDevice));

  // TODO: Set grid size and call kernel
  dim3 blockSize(TILE_WIDTH, TILE_WIDTH, 1);
  int W_grid = width_out / TILE_WIDTH;  // number of horizontal tiles per output map
  int H_grid = height_out / TILE_WIDTH; // number of vertical tiles per output map
  int Z_grid = H_grid * W_grid;
  dim3 gridSize(n_sample, channel_out, Z_grid);

  filter1<<<gridSize, blockSize>>>(d_in, channel_in, height_in, width_in,
                                     d_out, channel_out, height_out, width_out,
                                     d_filter, filterWidth, W_grid,
                                     d_bias);
  // Checks for synchronous errors
  hipError_t errSync = hipGetLastError();
  if (errSync != hipSuccess)
    printf("Sync kernel error: %s\n", hipGetErrorString(errSync));

  // TODO: Copy result from device memory
  CHECK(hipMemcpy(h_out, d_out, nBytes_d_out, hipMemcpyDeviceToHost));

  // TODO: Free device memories
  CHECK(hipFree(d_in));
  CHECK(hipFree(d_out));
  CHECK(hipFree(d_filter));
  CHECK(hipFree(d_bias));

  // hipDeviceReset(); // Force to print

  // return filter type
  return 1;
}

#endif